#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale (CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img) {
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y; // zdrojový obrázek
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_color_cuda_img.m_size.y) return;
    if (l_x >= t_color_cuda_img.m_size.x) return;

    // Get point from color picture
    uchar3 l_bgr = t_color_cuda_img.m_p_uchar3 [l_y * t_color_cuda_img.m_size.x + l_x];

    int height = t_color_cuda_img.m_size.y; // výška obrázku - (height - 1 - l_y)
    int width = t_color_cuda_img.m_size.x; // šířka obrázku - (width - 1 - l_x)

    // Store BW point to new image
    t_bw_cuda_img.m_p_uchar3 [l_y * t_bw_cuda_img.m_size.x + (width - 1 - l_x)] = l_bgr;
}

void cu_run_grayscale (CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img) {
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    dim3 l_blocks ((t_color_cuda_img.m_size.x + l_block_size - 1) / l_block_size, (t_color_cuda_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads (l_block_size, l_block_size);
    kernel_grayscale <<<l_blocks, l_threads>>> (t_color_cuda_img, t_bw_cuda_img);

    if (( l_cerr = hipGetLastError ()) != hipSuccess)
        printf ("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString (l_cerr));

    hipDeviceSynchronize ();
}

__global__ void kernel_color_level (CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img, uchar3 color) {
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y; // zdrojový obrázek
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_color_cuda_img.m_size.y) return;
    if (l_x >= t_color_cuda_img.m_size.x) return;

    // Get point from color picture
    // uchar3 l_bgr = t_color_cuda_img.m_p_uchar3 [l_y * t_color_cuda_img.m_size.x + l_x];
    uchar3 l_bgr = t_color_cuda_img.at3 (l_y, l_x);
    uchar3 finalPixel;

    if (color.x > l_bgr.x) {
        finalPixel.x = 0;
    } else finalPixel.x = l_bgr.x - color.x;
    
    if (color.y > l_bgr.y) {
        finalPixel.y = 0;
    } else finalPixel.y = l_bgr.y - color.y;

    if (color.z > l_bgr.z) {
        finalPixel.z = 0;
    } else finalPixel.z = l_bgr.z - color.z;

    // Store BW point to new image
    t_bw_cuda_img.m_p_uchar3 [l_y * t_bw_cuda_img.m_size.x + l_x].x = finalPixel.x;
    t_bw_cuda_img.m_p_uchar3 [l_y * t_bw_cuda_img.m_size.x + l_x].y = finalPixel.y;
    t_bw_cuda_img.m_p_uchar3 [l_y * t_bw_cuda_img.m_size.x + l_x].z = finalPixel.z;
}

void cu_run_color_level (CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img, uchar3 color) {
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    dim3 l_blocks ((t_color_cuda_img.m_size.x + l_block_size - 1) / l_block_size, (t_color_cuda_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads (l_block_size, l_block_size);
    kernel_color_level <<<l_blocks, l_threads>>> (t_color_cuda_img, t_bw_cuda_img, color); // spustím funkci na všech vláknech (samostatně)

    if ((l_cerr = hipGetLastError ()) != hipSuccess)
        printf ("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString (l_cerr));

    hipDeviceSynchronize ();
}